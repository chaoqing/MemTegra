#include <hip/hip_runtime.h>

#include <stdexcept>
#include <string>

#include "MemTegra/device_memory.h"

namespace MT {
    namespace internal {
        void* cuda_malloc(std::size_t size) {
            if (size == 0) {
                throw std::invalid_argument("Size must be greater than zero.");
            }

            void*       ptr = nullptr;
            hipError_t err = hipMalloc(&ptr, size);
            if (err != hipSuccess) {
                throw std::runtime_error("CUDA device memory allocation failed: "
                                         + std::string(hipGetErrorString(err)));
            }
            return ptr;
        }

        void cuda_free(void* ptr) {
            if (ptr != nullptr) {
                hipError_t err = hipFree(ptr);
                if (err != hipSuccess) {
                    throw std::runtime_error("CUDA device memory deallocation failed: "
                                             + std::string(hipGetErrorString(err)));
                }
            }
        }

    };  // namespace internal
};      // namespace MT
