#ifdef ENABLE_CUDA
#    include <hip/hip_runtime.h>
#else
#    include <cstring>
#endif

#include <cstdlib>
#include <iostream>
#include <stdexcept>
#include <string>

#include "MemTegra/device_memory.h"

namespace MT {
    namespace internal {
        void* cuda_malloc(std::size_t size) {
            if (size == 0) {
                throw std::invalid_argument("Size must be greater than zero.");
            }

            void* ptr = nullptr;
#ifdef ENABLE_CUDA
            if (::MT::cuda::is_available()) {
                hipError_t err = hipMalloc(&ptr, size);
                if (err != hipSuccess) {
                    throw std::runtime_error("CUDA device memory allocation failed: "
                                             + std::string(hipGetErrorString(err)));
                }
            } else
#endif
            {

                constexpr size_t CUDA_DEFAULT_ALIGNMENT = 256;
                ptr = std::aligned_alloc(CUDA_DEFAULT_ALIGNMENT, (size + CUDA_DEFAULT_ALIGNMENT - 1)
                                                                     / CUDA_DEFAULT_ALIGNMENT
                                                                     * CUDA_DEFAULT_ALIGNMENT);
            }
            if (ptr == nullptr) {
                throw std::bad_alloc();
            }
            return ptr;
        }

        void cuda_free(void* ptr) {
            if (ptr != nullptr) {
#ifdef ENABLE_CUDA
                if (::MT::cuda::is_available()) {
                    hipError_t err = hipFree(ptr);
                    if (err != hipSuccess) {
                        throw std::runtime_error("CUDA device memory deallocation failed: "
                                                 + std::string(hipGetErrorString(err)));
                    }
                } else
#endif
                {
                    std::free(ptr);
                }
            }
        }

    };  // namespace internal

    namespace cuda {
        namespace internal {
            bool _is_available() {
#ifdef ENABLE_CUDA
                int         device_count = 0;
                hipError_t err          = hipGetDeviceCount(&device_count);
                if (err != hipSuccess) {
                    return false;
                }
                return device_count > 0;
#else
                return false;
#endif
            }
        };  // namespace internal
        bool is_available() {
#ifdef ENABLE_CUDA
            static const bool state = internal::_is_available();
            return state;
#else
            return false;
#endif
        }

        std::pair<size_t, size_t> get_device_memory_usage() {
#ifdef ENABLE_CUDA
            if (::MT::cuda::is_available()) {
                size_t      free_mem, total_mem;
                hipError_t err = hipMemGetInfo(&free_mem, &total_mem);
                if (err != hipSuccess) {
                    throw std::runtime_error("Failed to get CUDA device memory usage: "
                                             + std::string(hipGetErrorString(err)));
                }
                return {free_mem, total_mem};
            }
#endif
            return {0, 0};
        }

        std::pair<int, int> device_get_stream_priority_range() {
#ifdef ENABLE_CUDA
            if (::MT::cuda::is_available()) {
                int         low_priority, high_priority;
                hipError_t err = hipDeviceGetStreamPriorityRange(&low_priority, &high_priority);
                if (err != hipSuccess) {
                    throw std::runtime_error("Failed to get CUDA stream priority range: "
                                             + std::string(hipGetErrorString(err)));
                }
                return {low_priority, high_priority};
            }
#endif
            return {0, 0};
        }
        void set_device_flags(cudaDeviceFlag flags) {
#ifdef ENABLE_CUDA

            if (::MT::cuda::is_available()) {
                using _Flag = decltype(hipDeviceScheduleAuto);
                static_assert(_Flag(cudaDeviceFlag::ScheduleAuto) == hipDeviceScheduleAuto);
                static_assert(_Flag(cudaDeviceFlag::ScheduleSpin) == hipDeviceScheduleSpin);
                static_assert(_Flag(cudaDeviceFlag::ScheduleYield) == hipDeviceScheduleYield);
                static_assert(_Flag(cudaDeviceFlag::ScheduleBlockingSync)
                              == hipDeviceScheduleBlockingSync);
                static_assert(_Flag(cudaDeviceFlag::ScheduleMask) == hipDeviceScheduleMask);
                static_assert(_Flag(cudaDeviceFlag::MapHost) == hipDeviceMapHost);
                static_assert(_Flag(cudaDeviceFlag::LmemResizeToMax) == hipDeviceLmemResizeToMax);
                static_assert(_Flag(cudaDeviceFlag::SyncMemops) == cudaDeviceSyncMemops);
                static_assert(_Flag(cudaDeviceFlag::Mask) == cudaDeviceMask);
                hipError_t err = hipSetDeviceFlags(_Flag(flags));
                if (err != hipSuccess) {
                    throw std::runtime_error("Failed to set CUDA device flags: "
                                             + std::string(hipGetErrorString(err)));
                }
            }
#endif
        }

        void host_register(void* ptr, size_t bytes, cudaHostRegisterFlag flags) {
#ifdef ENABLE_CUDA
            if (::MT::cuda::is_available()) {
                using _cudaFlag = decltype(hipHostRegisterDefault);

                static_assert(_cudaFlag(cudaHostRegisterFlag::Default) == hipHostRegisterDefault);
                static_assert(_cudaFlag(cudaHostRegisterFlag::Portable)
                              == hipHostRegisterPortable);
                static_assert(_cudaFlag(cudaHostRegisterFlag::Mapped) == hipHostRegisterMapped);
                static_assert(_cudaFlag(cudaHostRegisterFlag::IoMemory)
                              == hipHostRegisterIoMemory);
                static_assert(_cudaFlag(cudaHostRegisterFlag::ReadOnly)
                              == hipHostRegisterReadOnly);
                hipError_t err = hipHostRegister(ptr, bytes, _cudaFlag(flags));
                if (err != hipSuccess) {
                    throw std::runtime_error("Failed to register host memory: "
                                             + std::string(hipGetErrorString(err)));
                }
            }
#endif
        }

        void* context::cuda_memset(void* dest, int ch, size_t n) const {
#ifdef ENABLE_CUDA
            if (::MT::cuda::is_available()) {
                hipError_t err = hipSuccess;
                if (async_) {
                    err = hipMemsetAsync(dest, ch, n, static_cast<::hipStream_t>(stream_));
                } else {
                    err = hipMemset(dest, ch, n);
                }
                if (err != hipSuccess) {
                    throw std::runtime_error("CUDA device memory memset failed: "
                                             + std::string(hipGetErrorString(err)));
                }
            } else
#endif
            {
                std::memset(dest, ch, n);
            }

            return dest;
        }

        void* context::cuda_memcpy(void* dest, const void* src, size_t n,
                                   hipMemcpyKind kind) const {
#ifdef ENABLE_CUDA
            if (::MT::cuda::is_available()) {
                hipError_t err = hipSuccess;

                static_assert(static_cast<::hipMemcpyKind>(hipMemcpyKind::hipMemcpyHostToHost)
                              == hipMemcpyHostToHost);
                static_assert(static_cast<::hipMemcpyKind>(hipMemcpyKind::hipMemcpyHostToDevice)
                              == hipMemcpyHostToDevice);
                static_assert(static_cast<::hipMemcpyKind>(hipMemcpyKind::hipMemcpyDeviceToHost)
                              == hipMemcpyDeviceToHost);
                static_assert(
                    static_cast<::hipMemcpyKind>(hipMemcpyKind::hipMemcpyDeviceToDevice)
                    == hipMemcpyDeviceToDevice);

                const auto _kind = static_cast<::hipMemcpyKind>(kind);
                if (async_) {
                    err = hipMemcpyAsync(dest, src, n, _kind,
                                          static_cast<::hipStream_t>(stream_));
                } else {
                    err = hipMemcpy(dest, src, n, _kind);
                }

                if (err != hipSuccess) {
                    throw std::runtime_error("CUDA device memory memset failed: "
                                             + std::string(hipGetErrorString(err)));
                }
            } else
#endif
            {
                std::memcpy(dest, src, n);
            }

            return dest;
        }

        std::unique_ptr<context> context::new_with_priority(int priority, bool async) {
#ifdef ENABLE_CUDA
            if (::MT::cuda::is_available()) {
                ::hipStream_t _stream;
                hipError_t    err
                    = hipStreamCreateWithPriority(&_stream, hipStreamDefault, priority);
                if (err != hipSuccess) {
                    throw std::runtime_error("Failed to create CUDA stream with priority: "
                                             + std::string(hipGetErrorString(err)));
                }
                auto c     = std::make_unique<context>(async);
                c->stream_ = _stream;

                return std::move(c);
            }
#endif
            return std::make_unique<context>();
        }


        void context::release() {
#ifdef ENABLE_CUDA
            if (stream_ != nullptr) {
                if (::MT::cuda::is_available()) {
                    hipError_t err = hipStreamDestroy(static_cast<::hipStream_t>(stream_));
                    stream_         = nullptr;  // avoid double throw even if it fail to release
                    if (err != hipSuccess) {
                        throw std::runtime_error("Failed to release the related stream: "
                                                 + std::string(hipGetErrorString(err)));
                    }
                }
            }
#endif
        }

        context::~context() {
            try {
                release();
            } catch (const std::exception& e) {
                std::cerr << "Exception in context destructor: " << e.what() << std::endl;
            } catch (...) {
                std::cerr << "Unknown exception in context destructor." << std::endl;
            }
        }

        void context::synchronize() {
#ifdef ENABLE_CUDA
            if (::MT::cuda::is_available()) {
                hipError_t err = hipStreamSynchronize(static_cast<::hipStream_t>(stream_));
                if (err != hipSuccess) {
                    throw std::runtime_error("Failed to synchronize CUDA stream: "
                                             + std::string(hipGetErrorString(err)));
                }
            }
#endif
        }

    };  // namespace cuda

};  // namespace MT
