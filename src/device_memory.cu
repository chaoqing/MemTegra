#ifdef ENABLE_CUDA
#    include <hip/hip_runtime.h>
#else
#    include <cstring>
#endif

#include <cstdlib>
#include <stdexcept>
#include <string>

#include "MemTegra/device_memory.h"

namespace MT {
    namespace internal {
        void* cuda_malloc(std::size_t size) {
            if (size == 0) {
                throw std::invalid_argument("Size must be greater than zero.");
            }

            void* ptr = nullptr;
#ifdef ENABLE_CUDA
            hipError_t err = hipMalloc(&ptr, size);
            if (err != hipSuccess) {
                throw std::runtime_error("CUDA device memory allocation failed: "
                                         + std::string(hipGetErrorString(err)));
            }
            return ptr;
#else
            constexpr size_t CUDA_DEFAULT_ALIGNMENT = 256;
            return std::aligned_alloc(size, CUDA_DEFAULT_ALIGNMENT);
#endif
        }

        void cuda_free(void* ptr) {
            if (ptr != nullptr) {
#ifdef ENABLE_CUDA
                hipError_t err = hipFree(ptr);
                if (err != hipSuccess) {
                    throw std::runtime_error("CUDA device memory deallocation failed: "
                                             + std::string(hipGetErrorString(err)));
                }
#else
                std::free(ptr);
#endif
            }
        }

        void* cuda_memset(void* dest, int ch, size_t n, hipStream_t stream) {
#ifdef ENABLE_CUDA
            hipError_t err = hipSuccess;
            if (stream) {
                err = hipMemsetAsync(dest, ch, n, static_cast<::hipStream_t>(stream));
            } else {
                err = hipMemset(dest, ch, n);
            }
            if (err != hipSuccess) {
                throw std::runtime_error("CUDA device memory memset failed: "
                                         + std::string(hipGetErrorString(err)));
            }
#else
            std::memset(dest, ch, n);
#endif

            return dest;
        }

        void* cuda_memcpy(void* dest, const void* src, size_t n, hipMemcpyKind kind,
                          hipStream_t stream) {
#ifdef ENABLE_CUDA
            hipError_t err = hipSuccess;

            static_assert(static_cast<::hipMemcpyKind>(hipMemcpyKind::hipMemcpyHostToDevice)
                          == hipMemcpyHostToDevice);
            static_assert(static_cast<::hipMemcpyKind>(hipMemcpyKind::hipMemcpyDeviceToHost)
                          == hipMemcpyDeviceToHost);
            static_assert(static_cast<::hipMemcpyKind>(hipMemcpyKind::hipMemcpyDeviceToDevice)
                          == hipMemcpyDeviceToDevice);

            const auto _kind = static_cast<::hipMemcpyKind>(kind);
            if (stream) {
                err = hipMemcpyAsync(dest, src, n, _kind, static_cast<::hipStream_t>(stream));
            } else {
                err = hipMemcpy(dest, src, n, _kind);
            }

            if (err != hipSuccess) {
                throw std::runtime_error("CUDA device memory memset failed: "
                                         + std::string(hipGetErrorString(err)));
            }
#else
            std::memcpy(dest, src, n);
#endif

            return dest;
        }

    };  // namespace internal
};      // namespace MT
